//compile with
//nvcc delta.cu -I /home/baggio/NVIDIA_CUDA_SDK/common/inc/ -L /home/baggio/NVIDIA_CUDA_SDK/lib/ -lcuda -lcudart -lcutil -lGL -lGLU
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

// includes, project
#include <cutil.h>
#include <cuda_gl_interop.h>
//#define TIMER

// includes, kernels
#include <delta_kernel.cu>



#define INF 1e20


float* pixels = NULL;
hipArray *array = NULL;

struct edge{
  int dNode[4];
  float weight[4];
};

edge* nodes;





void
runTest( int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{

    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}

void loadTexture(int iw, int ih, float* data){
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL(hipMallocArray(&array, &desc, iw, ih));
    CUDA_SAFE_CALL(hipMemcpyToArray(array, 0, 0, data, sizeof(float)*iw*ih, hipMemcpyHostToDevice));
    // Bind the array to the texture
    hipBindTextureToArray( tex, array, desc);

}

void loadGraphEdges(){
  int n,m;
  FILE* in = fopen("512.graph","r");//myFile,"r");
  fscanf(in,"%d\n",&n);
  
  while(1){
    int source, dest;
    double eWeight;
    fscanf(in,"%d",&source);
//    printf("Reading %d\n",source);
    if(source==-1) break;
    fscanf(in,"%d%lf\n",&dest,&eWeight);

    nodes[source].dNode[0]= dest;
    nodes[source].weight[0]= (float)eWeight;

    nodes[dest].dNode[0] = source;
    nodes[dest].weight[0] = (float)eWeight;

  }
  
}


void loadGraph(int imgWidth,int imgHeight){

    pixels = (float*) malloc (imgWidth*imgHeight*sizeof(float));
    for(int i=0;i<imgWidth*imgHeight;i++){
        pixels[i]=nodes[i].weight[0];//(2*i);
//        printf("Loading edge %f real %f\n",nodes[i].weight[0],pixels[i]);
    }
    loadTexture(imgWidth,imgHeight,pixels);

}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{



    //initialize the device
    hipSetDevice(0);

    unsigned int num_threads = 256;//512;
//    unsigned int mem_size = sizeof( float) * num_threads;

    // setup execution parameters
    dim3  grid( 1, 1, 1);
    dim3  threads( num_threads, 1, 1);
    int iw=512, ih=512;

    nodes = (edge*) malloc(GN*sizeof(edge));
//	printf("Before loading\n");   
    loadGraphEdges();
//	printf("After loading\n");
    loadGraph(iw,ih);
//	printf("After loading\n");
    //allocate Bucket memory






    float* dDist;
    hipMalloc( (void**) &dDist, GN*sizeof(float));
//    float hDist[Gn];
    float* hDist = (float*) malloc(GN*sizeof(float));
    
    for(int i=0;i<GN;i++){
      hDist[i]=INF;
    }
    for(int i=0;i<200;i++){
      hDist[i]=0.0;
    }

    hipMemcpy( dDist, hDist, GN*sizeof(float), hipMemcpyHostToDevice);

    int* dBucketMap;
    hipMalloc( (void**) &dBucketMap, GN*sizeof(int));
    int* hBucketMap = (int*) malloc(GN*sizeof(int));
    
     
    for(int i=0;i<GN;i++){
      hBucketMap[i]=-1;
    }

    hipMemcpy( dBucketMap, hBucketMap, GN*sizeof(int), hipMemcpyHostToDevice);

    int* dBucketPos;
    hipMalloc( (void**) &dBucketPos, GN*sizeof(int));
    int* hBucketPos = (int*) malloc(GN*sizeof(int));

    int* dB;
    hipMalloc( (void**) &dB, BUCKETSIZE*NUMBUCKETS*sizeof(int));
    int* hB = (int*) malloc(BUCKETSIZE*NUMBUCKETS*sizeof(int));
  

    printf("Bucket size %d\n",BUCKETSIZE);

    int* hBi = hB;
    int hBiCount[1];
    hBiCount[0] = 200;
    for(int i=0;i<BUCKETSIZE*NUMBUCKETS;i++)
      hBi[i]=-1;
    for(int i=0;i<hBiCount[0];i++)
      hBi[i]=i;

    //duplicated values to test
//    hBi[0]=8;
//    hBi[31]=8;



    // copy host memory to device
    hipMemcpy( dB, hBi, BUCKETSIZE*NUMBUCKETS*sizeof(int), hipMemcpyHostToDevice);


    int* BCount;
    hipMalloc( (void**) &BCount, NUMBUCKETS * sizeof(int));
    int* hBCount;
    hBCount = (int*) malloc(NUMBUCKETS*sizeof(int));
    for(int i=0;i<NUMBUCKETS;i++)
      hBCount[i]=0;

    hBCount[0]=200;

    // copy host memory to device
    hipMemcpy( BCount, hBCount, NUMBUCKETS*sizeof(int), hipMemcpyHostToDevice);

    // allocate device memory for result
    int * dRLoc;
    hipMalloc( (void**) &dRLoc, GN*sizeof(int));
    int * dR;
    hipMalloc( (void**) &dR, 4*8*BUCKETSIZE*sizeof(int)); //R is 4 times bigger than RLoc, because UP,DOWN,LEFT and RIGHT will each have a pos in R

    int* hR = (int*) malloc(4*8*BUCKETSIZE*sizeof(int));
    int* hRLoc = (int*) malloc(GN*sizeof(int));
    float* hDistR= (float*) malloc(4*8*BUCKETSIZE*sizeof(float));


    float* dDistR;
    hipMalloc( (void**) &dDistR, 4*8*BUCKETSIZE*sizeof(float));


    int * dS;
    hipMalloc( (void**) &dS, 8*BUCKETSIZE*sizeof(int));
    int * hS= (int*) malloc(8*BUCKETSIZE*sizeof(int));
    int * dSCount;
    hipMalloc( (void**) &dSCount, 1*sizeof(int));
    int * hSCount= (int*) malloc(1*sizeof(int));

    printf("Starting timer\n");

  unsigned int nvtimer = 0;
    cutCreateTimer( &nvtimer);
    cutStartTimer( nvtimer);


  unsigned int laptimer = 1;
    cutCreateTimer( &laptimer);
  float ktime;

       float* lido;
    hipMalloc( (void**) &lido, GN*sizeof(float));
     float* Hlido = (float*) malloc(GN*sizeof(float)) ;
    


  
    for(int i=0;i<1024;i++){
#ifdef TIMER
      printf("%d\n",i);
#endif
      int* RCount;
      RCount = (int*) malloc(1*sizeof(int));
      hipMemcpy( RCount,  &BCount[i],  1*sizeof(int),       hipMemcpyDeviceToHost) ;    


      //S <- EMPTY
      hSCount[0]=0;
      hipMemcpy( dSCount, hSCount,1*sizeof(int), hipMemcpyHostToDevice);
    
//  printf("Before RCount %d(i=%d)\n",RCount[0],i);
  //    printf("Still safe%d\n",i);

      //While B[i] != EMPTY
      int sameCount = 0;
      cutStartTimer( laptimer);
      while(RCount[0]!=0){
//      if(RCount[0]==0) i++;
        sameCount++;

#ifdef TIMER
        cutStartTimer( laptimer);
#endif

       

        labelKernelDebug    <<<grid, threads >>> ( i, dB, BCount, iw, dRLoc, dR, dDistR, dDist,dBucketMap,lido);
        hipDeviceSynchronize();
        labelKernelDebug1   <<<grid, threads >>> ( i, dB, BCount, iw, dRLoc, dR, dDistR, dDist,dBucketMap,lido);

//        labelKernel    <<<grid, threads >>> ( i, dB, BCount, iw, dRLoc, dR, dDistR, dDist,dBucketMap);
        hipDeviceSynchronize();

if(i>100){
	hipMemcpy( hB, dB, BUCKETSIZE*NUMBUCKETS*sizeof(int),hipMemcpyDeviceToHost);
	        for(int j=0;j<RCount[0];j++){ 
		  if(hB[BUCKETSIZE*i+j]%512>=200){
		  printf("Alarm!!! B[%d]=%d (i=%d)\n",j,hB[BUCKETSIZE*i+j],i);
 		  }
	        }
	int* mytcount = (int*) malloc(1*sizeof(int));
	hipMemcpy( mytcount,  &BCount[i+1],  1*sizeof(int),       hipMemcpyDeviceToHost) ;    
	        for(int j=0;j<mytcount[0];j++){ 
		  if(hB[BUCKETSIZE*(i+1)+j]%512>=200){
		  printf("Alarm(lookahead)!!! B[%d]=%d (i=%d)\n",j,hB[BUCKETSIZE*(i+1)+j],i+1);
 		  }
	        }

}




#ifdef TIMER
        ktime = cutGetTimerValue( laptimer );
        printf("Label         kernel %f\n",ktime);	
        cutStartTimer( laptimer);
#endif

        copyB2SKernel  <<<grid, threads >>> ( i, dB, BCount, dS, dSCount);
        hipDeviceSynchronize();


if(i>100){
      int mytemp[1];
      hipMemcpy( mytemp,  &dSCount[0],  1*sizeof(int),       hipMemcpyDeviceToHost) ;

	hipMemcpy( hS, dS,8*BUCKETSIZE*sizeof(int),hipMemcpyDeviceToHost);
	        for(int j=0;j<mytemp[0];j++){ 
		  if(hS[j]%512>=200){
		  printf("Heavyalarm!!! S[%d]=%d (i=%d)\n",j,hS[j],i);
 		  }
		  else{
                    printf("S[%d]=%d (i=%d)\n",j,hS[j],i);
		  }
	        }
}


#ifdef TIMER
        ktime = cutGetTimerValue( laptimer );
        printf("CopyB2S       kernel %f\n",ktime);	
#endif

#ifdef TIMER
        cutStartTimer( laptimer);
#endif

        emptyKernel    <<<grid, threads >>> ();

#ifdef TIMER
        ktime = cutGetTimerValue ( laptimer );
	printf(" Empty        kernel %f\n",ktime);
#endif

        //todo: correct RCount
        cutStartTimer( laptimer);

        relaxKernel    <<<grid, threads >>> ( RCount[0], dB, BCount, dRLoc, dR, dDistR, dDist,dBucketPos, dBucketMap,lido);
        hipDeviceSynchronize();

#ifdef TIMER
        ktime = cutGetTimerValue( laptimer );
        printf("Relaxing      kernel %f\n",ktime);
#endif


        hipMemcpy( RCount,  &BCount[i],  1*sizeof(int),       hipMemcpyDeviceToHost) ;
        hipMemcpy( hSCount,  dSCount,  1*sizeof(int),       hipMemcpyDeviceToHost) ;        

       printf("After RCount %d (i=%d) | SCount %d\n",RCount[0],i,hSCount[0]);

	

#ifdef TIMER
        printf("After RCount %d (i=%d) | SCount %d\n",RCount[0],i,hSCount[0]);
#endif


      }

//      ktime = cutGetTimerValue( laptimer );
#ifdef TIMER
      printf("Same called %d times.\n",sameCount);
#endif

//      printf("Label kernel %f\n",ktime);	
// 	printf("Before heavy label%d\n",i);      

	printf("Before heavy relaxing\n");

if(i>100){
      int mytemp[1];
      hipMemcpy( mytemp,  &dSCount[0],  1*sizeof(int),       hipMemcpyDeviceToHost) ;

	hipMemcpy( hS, dS,8*BUCKETSIZE*sizeof(int),hipMemcpyDeviceToHost);
	        for(int j=0;j<mytemp[0];j++){ 
		  if(hS[j]%512>=200){
		  printf("before heavy label Heavyalarm!!! S[%d]=%d (i=%d)\n",j,hS[j],i);
 		  }
		  else{
                    printf("before heavy label S[%d]=%d (i=%d)\n",j,hS[j],i);
		  }
	        }
}

      cutStartTimer( laptimer);
      labelHeavyKernel    <<<grid, threads >>> ( i, dS, dSCount, iw, dRLoc, dR, dDistR, dDist,dBucketMap);
      hipDeviceSynchronize();

if(i>100){
      int mytemp[1];
      hipMemcpy( mytemp,  &dSCount[0],  1*sizeof(int),       hipMemcpyDeviceToHost) ;

	hipMemcpy( hR, dR,4*8*BUCKETSIZE*sizeof(int),hipMemcpyDeviceToHost);
	        for(int j=0;j<mytemp[0];j++){ 
		  if(hR[j]%512>=200){
		  printf("heavylab Heavyalarm!!! R[%d]=%d (i=%d)\n",j,hR[j],i);
 		  }
		  else{
                    printf("heavylab R[%d]=%d (i=%d)\n",j,hR[j],i);
		  }
	        }
}




      ktime = cutGetTimerValue( laptimer );
#ifdef TIMER
      printf("Labelling heavy kernel %f\n",ktime);
#endif

// 	printf("After heavy label%d\n",i);      
      //todo: correct SCount

#ifdef TIMER
      cutStartTimer( laptimer);
#endif
      int temp[1];
      hipMemcpy( temp,  &dSCount[0],  1*sizeof(int),       hipMemcpyDeviceToHost) ;
      relaxKernel         <<<grid, threads >>> ( temp[0], dB, BCount, dRLoc, dR, dDistR, dDist,dBucketPos, dBucketMap,lido);
      ktime = cutGetTimerValue( laptimer );
#ifdef TIMER
      printf("Relaxing heavy  kernel %f\n",ktime);
#endif

if(i>100){
	hipMemcpy( hB, dB, BUCKETSIZE*NUMBUCKETS*sizeof(int),hipMemcpyDeviceToHost);
	        for(int j=0;j<RCount[0];j++){ 
		  if(hB[BUCKETSIZE*i+j]%512>=200){
		  printf("hrelax Alarm!!! B[%d]=%d (i=%d)\n",j,hB[BUCKETSIZE*i+j],i);
 		  }
	        }
	int* mytcount = (int*) malloc(1*sizeof(int));
	hipMemcpy( mytcount,  &BCount[i+1],  1*sizeof(int),       hipMemcpyDeviceToHost) ;    
	        for(int j=0;j<mytcount[0];j++){ 
		  if(hB[BUCKETSIZE*(i+1)+j]%512>=200){
		  printf("hrelax Alarm(lookahead)!!! B[%d]=%d (i=%d)\n",j,hB[BUCKETSIZE*(i+1)+j],i+1);
 		  }
	        }

}



	


// 	printf("After heavy relax%d\n",i);      

//      printf("Done here. i = %d\n",i);


    }
    // check if kernel execution generated and error

   CUT_CHECK_ERROR("Kernel execution failed");

 

    ktime = cutGetTimerValue( nvtimer );
    hipMemcpy( hRLoc,  dRLoc,  1024*sizeof(int),       hipMemcpyDeviceToHost) ;    
    hipMemcpy( hRLoc,  dRLoc,  1024*sizeof(int),       hipMemcpyDeviceToHost) ;
    hipMemcpy( hR,     dR,     4*1024*8*sizeof(int),   hipMemcpyDeviceToHost) ;
    hipMemcpy( hDistR, dDistR, 4*1024*8*sizeof(float), hipMemcpyDeviceToHost) ;
    hipMemcpy( hS,     dS,     1024*8*sizeof(float),   hipMemcpyDeviceToHost) ;
    hipMemcpy( hDist,  dDist,  GN*sizeof(int),       hipMemcpyDeviceToHost) ;
    hipMemcpy( hBCount, BCount,  NUMBUCKETS*sizeof(int),       hipMemcpyDeviceToHost) ;
    hipMemcpy( hB, dB,           2*NUMBUCKETS*sizeof(int),       hipMemcpyDeviceToHost) ;

	hipMemcpy( Hlido, lido, GN*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy( hR, dR, GN*sizeof(int),hipMemcpyDeviceToHost);
        for(int j=0;j<GN;j++){ 
	  printf("lido[%d]=%f\n",j,Hlido[j]);
        }



      for(int j=0;j<200;j++){

        printf("R[%d] %d dR[%d] %f RLoc[%d]=%d\n",j,hR[j],j,hDistR[j],j,hRLoc[j]);
      }
    for(int i=0;i<512*512;i++){
      if(i%512==0) printf("%3d",i/512);
      if(hDist[i]<INF)
        printf("%5.1f ",hDist[i]);
      else
        printf("INFINI ",hDist[i]);
      if(i%512==511) printf("\n");
    }      


  /*  printf("Num threads %d\n",num_threads);
    for(int i=0;i< 20;i++){
      printf("RLoc[%d] = %d\n",i,hRLoc[i]);
    }
    for(int i=0;i< 33;i++){
      printf("R[%d] = %d (%f)\n",i,hR[i],hDistR[i]);
    }
    for(int i=0;i< 20;i++){
      printf("S[%d] = %d \n",i,hS[i]);
    }
    for(int i=0;i< 128;i++){
      printf("d[%d] = %f \n",i,hDist[i]);
    }
    printf("BiCount %d\n",hBiCount[0]);
    
    for(int i=0;i< 20;i++){
      printf("hBCount[%d] = %d \n",i,hBCount[i]);
    }

    for(int i=0;i< 20;i++){
      printf("hB[%d] = %d \n",i,hB[i]);
    }
    for(int i=BUCKETSIZE;i<BUCKETSIZE+ 20;i++){
      printf("hB[%d] = %d \n",i,hB[i]);
    }
*/




    printf("It took %f ms\n", ktime);

    hipFreeArray(array);
    hipFree(dB);
    hipFree(dRLoc);
    hipFree(dDist);
    hipFree(dBucketMap);
    hipFree(dBucketPos);

    hipFree(BCount);

    hipFree(dR);
    hipFree(dDistR);
    hipFree(dS);
    hipFree(dSCount);
  




  /*




    // allocate host memory
    float* h_idata = (float*) malloc( mem_size);

    // initalize the memory
    for( unsigned int i = 0; i < num_threads ; ++i) 
    {
        h_idata[i] = (float) i;
    }


    //float initTime = cutGetTimerValue( timer );

//    printf("Host init time %f ms\n", initTime);
    printf("Here\n");
    unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    cutStartTimer(timer);



    // allocate device memory
    float* d_idata;
    hipMalloc( (void**) &d_idata, mem_size);


    float initTime = cutGetTimerValue( timer );
    printf("Allocate device memory time %f ms\n", initTime);
    cutStartTimer(timer);

    // copy host memory to device
    hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) ;

    initTime = cutGetTimerValue( timer );
    printf("Copy host to device time %f ms\n", initTime);
    cutStartTimer(timer);


    // allocate device memory for result
    float* d_odata;
    hipMalloc( (void**) &d_odata, mem_size);

    initTime = cutGetTimerValue( timer );
    printf("Allocate device memory for result time %f ms\n", initTime);
    cutStartTimer(timer);


   


    // execute the kernel
    deltaKernel<<< grid, threads, mem_size >>>( d_idata, d_odata);

    initTime = cutGetTimerValue( timer );
    printf("Executing kernel time %f ms\n", initTime);
    cutStartTimer(timer);


    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( mem_size);
    // copy result from device to host
    hipMemcpy( h_odata, d_odata, sizeof( float) * num_threads,
                                hipMemcpyDeviceToHost) ;

    initTime = cutGetTimerValue( timer );
    printf("Readback time %f ms\n", initTime);
    cutStartTimer(timer);

    cutStopTimer( timer);
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    cutDeleteTimer( timer);

    for(int i=0;i< num_threads;i++){
      printf("%f\n",h_odata[i]);
    } 
    // cleanup memory
    free( h_idata);
    free( h_odata);
    hipFree(d_idata);
    hipFree(d_odata);*/
}

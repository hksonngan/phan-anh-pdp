#include "hip/hip_runtime.h"
#ifndef _DELTA_KERNEL_H_
#define _DELTA_KERNEL_H_

#include <stdio.h>
#define DELTA 1.0
#define INF 1e20
#define GN 262144
#define BUCKETSIZE 4096*8 //era 4096*8
#define NUMBUCKETS 1024

#define DOWN  0
#define UP    1
#define RIGHT 2
#define LEFT  3


//#define EMULATION

#ifdef EMULATION
#define DEBUG(x...) printf(x)
#else
#define DEBUG(x...) 
#endif

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
texture<float, 2, hipReadModeElementType> mytex0;
texture<float, 2, hipReadModeElementType> mytex1;
texture<float, 2, hipReadModeElementType> mytex2;
texture<float, 2, hipReadModeElementType> mytex3;

__device__ void
demptyKernel(){
}

/*
__global__ void
deltaKernel( float* g_idata, float* g_odata) 
{


  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  //  const unsigned int num_threads = blockDim.x;

  // shared memory
  // the size is determined by the host application
  extern __shared__  float sdata[];


  // read in input data from global memory
  sdata[tid] = tex2D(tex,2,2);
  __syncthreads();

  // perform some computations
  sdata[tid] = (float) sdata[tid];
  __syncthreads();

  // write data to global memory
  g_odata[tid] = sdata[tid];
}
*/

__global__ void
labelKernel (int i, int* B,int* BCount,int* BPos, int tw,int th, int* RLoc,int* R, float* dR, float* d,int* vBucketMap){
//ver se não precisa colocar 4 BiCount em algum lugar...

  //todo: try to increase speed using shared memory for RLoc... think more about it (maybe RLoc is too big for shared memory)
  const unsigned int tid = threadIdx.x;
  const unsigned int num_threads = blockDim.x;

  int BiCount = BPos[i];
  int node, row, col, pos,index;
  float cost,f1,f2,fmin;

  //cleaning R
  //4 times because each node can be reached from up, down, left and right directions (and more 4 times because for each node 4 more are open)
//  DEBUG("BiCount %d lastpos %d\n",BiCount,4*(BiCount-1)+3);
  for(int k=0;  (num_threads*k + tid) < 16*BiCount;k++){
    index = 4*(num_threads*k + tid);
    R[index  ]=-1;
    R[index+1]=-1;
    R[index+2]=-1;
    R[index+3]=-1;
  
    dR[index  ]=INF;
    dR[index+1]=INF;
    dR[index+2]=INF;
    dR[index+3]=INF;
  }

  __syncthreads();
  const int dx[4]={0,0,1,-1};
  const int dy[4]={1,-1,0,0};


/*  //down direction
  dx[DOWN]=  0;
  dy[DOWN]=  1;

  //up direction
  dx[UP]=  0;
  dy[UP]= -1;

  //right direction
  dx[RIGHT]=  1;
  dy[RIGHT]=  0;

  //left direction
  dx[LEFT]= -1;
  dy[LEFT]=  0;*/
  

  for(int k=0; num_threads*k + tid < BiCount;k++){
    node = B  [ BUCKETSIZE*i + num_threads*k + tid];
    if(node!=-1){

      DEBUG("(tid %d) node %d(from B[%d] pos %d)\n",tid,node,i,num_threads*k + tid);
      for(int j=0;j<4;j++){
        //TRY TO OPTIMIZE THIS PART REMOVING THE IF AND SETTING IMPOSSIBLE EDGE VALUES TO INFINITY
        switch(j){
	  case 0:
           cost = tex2D(mytex0,node%tw,node/tw);
           break;
	  case 1:
           cost = tex2D(mytex1,node%tw,node/tw);
           break;
	  case 2:
           cost = tex2D(mytex2,node%tw,node/tw);
           break;
	  case 3:
           cost = tex2D(mytex3,node%tw,node/tw);
           break;
        }
//      cost = tex2D(mytex[j],node%tw,node/tw);//todo: change texture
        row = node/tw + dy[j];
        col = node%tw + dx[j];

        if( (row>=0) && (row < th) && (col >= 0) && (col < tw) ){
          RLoc [ row*tw + col ] = 4*(num_threads*k + tid)+j;
          DEBUG("(tid %d)Connecting node %d to be processed by %d\n",tid,row*tw+col,RLoc [ row*tw + col ]);
        }

      }

    }
  }

  __syncthreads();

  //copy Edges to R
  for(int k=0; num_threads*k + tid < BiCount;k++){
    node = B  [ BUCKETSIZE*i + num_threads*k + tid];
    if(node!=-1){
      for(int j=0;j<4;j++){
        switch(j){
  	case 0:
         cost = tex2D(mytex0,node%tw,node/tw);
         break;
	case 1:
         cost = tex2D(mytex1,node%tw,node/tw);
         break;
	case 2:
         cost = tex2D(mytex2,node%tw,node/tw);
         break;
	case 3:
         cost = tex2D(mytex3,node%tw,node/tw);
         break;
      }



        row = node/tw + dy[j];
        col = node%tw + dx[j];
              DEBUG("Pre-candidate in R %d (d=%f)\n",row*tw + col,d[node]+cost,dR [4*RLoc[row*tw + col]+j]);
        if( (row>=0) && (row < th) && (col >= 0) && (col < tw) ){ 
            DEBUG("Candidate in R %d (d=%f)\n",row*tw + col,d[node]+cost,dR [4*RLoc[row*tw + col]+j]);
          if((cost<=DELTA)&&( d[node]+cost < d[row*tw + col])){
              R [4*RLoc[row*tw + col]+j] = row*tw + col;
              dR[4*RLoc[row*tw + col]+j] = d[node]+cost;
              DEBUG("New node in R %d (d=%f,e=%f) in pos %d\n",R [4*RLoc[row*tw + col]+j],dR [4*RLoc[row*tw + col]+j],cost,4*RLoc[row*tw + col]+j);
              vBucketMap[node]=-1;
          }
        }
      }
    }
  }
  __syncthreads();

  //gathering data to find the minimum cost way to get to node n
  //TODO: OPTIMIZE IN SUCH A WAY IT WON'T BE NEEDED TO GO THROUGH THE 4 EDGES, since they store the same value
  for(int k=0;  (num_threads*k + tid) < 16*BiCount;k++){
    f1 = fminf( dR[4*(num_threads*k + tid)  ], dR[4*(num_threads*k + tid)+1] );
    f2 = fminf( dR[4*(num_threads*k + tid)+2], dR[4*(num_threads*k + tid)+3] );
    fmin = fminf(f1,f2);
    dR[4*(num_threads*k + tid)  ]=fmin;
    dR[4*(num_threads*k + tid)+1]=fmin;
    dR[4*(num_threads*k + tid)+2]=fmin;
    dR[4*(num_threads*k + tid)+3]=fmin;
  }
  __syncthreads();


//  demptyKernel();
//  __syncthreads();

  
}
__global__ void 
copyB2SKernel(int i, int* B, int* BCount,int* BPos, int* S, int* SCount){
  //TODO: optimize this code
  //there's an optimized way of doing this, which is by only
  //storing SCount = Scount+ BCount, as output
  //and controlling with local variables thread positions

  const unsigned int tid = threadIdx.x;
  const unsigned int num_threads = blockDim.x;

  int pos;

  int BiCount = BCount[i];
  for(int k=0; num_threads*k + tid < BiCount;k++){
    if(B[i*BUCKETSIZE+num_threads*k+tid]!=-1){
      pos = atomicAdd(&SCount[0],1);
      S[pos] = B[i*BUCKETSIZE+num_threads*k+tid];
    }
  }
  __syncthreads();
  BCount[i]=0;
  BPos[i]=0;
  __syncthreads();
//  DEBUG("(tid %d) SCount %d\n",tid,SCount[0]);


}

//Parallel relax edges
__global__ void
relaxKernel( int RCount, int* B,int* BCount,int* BPos, int* RLoc,int* R,float*  dR,float* d, int* vBucketLoc, int* vBucketMap, float* deb){

  const unsigned int tid = threadIdx.x;
  const unsigned int num_threads = blockDim.x;
  int v,bn,bn_old, index;
  float x;
//  int myAdd=0;
  deb[20]= (float)RCount;

  for(int k=0; num_threads*k + tid < RCount;k++){
    index = num_threads*k + tid;
    deb[2*index]= R[index];
    deb[2*index+1]= dR[index];
  }

//  DEBUG("relaxing RCount %d\n",RCount);

  //remove node from old bucket
//  RCount = RCount /4;



  for(int k=0; num_threads*k + tid < RCount;k++){
    index = num_threads*k + tid;
  //  deb[index]= R[index];

    if(R[index]!=-1){

      x = dR[index];      
      v = R[index];

      if(x<d[v]){

        bn_old = vBucketMap[v];
        if (bn_old != -1) {
          int oldIndex = bn_old*BUCKETSIZE+vBucketLoc[v];
//	  
          B[oldIndex] = -1;//GN;
          int oldc = atomicSub(&BCount[bn_old],1);

//        printf("Removing %d from %d(%d)\n",v,bn_old,oldc);


        }
      }

    }    

  }




  __syncthreads();






  for(int k=0; num_threads*k + tid < RCount;k++){

    if(R[num_threads*k + tid]!=-1){
//deb[0]= (float) (BUCKETSIZE);
      x = dR[num_threads*k + tid];
      v = R[num_threads*k + tid];
      if(x < d[v]){



      bn = (int) (dR[num_threads*k + tid]/DELTA); 
      
   //   printf("Bn %d\n",bn);

      atomicAdd(&BCount[bn],1);
      int pos = atomicAdd(&BPos[bn],1);
      DEBUG("Pos %d BCount[%d] %d node %d (x=%f)\n",pos,bn,BPos[bn],v,x);

      
      B[bn*BUCKETSIZE+pos] = v;
      d[v] = x;

      
      vBucketLoc[v] = pos;  
      vBucketMap[v] = bn;
      RLoc[v]=-1;


  //only debug info
//    for(int i=0;i<BPos[bn];i++){
//      DEBUG("B(%d)=%d ",i,B[bn*BUCKETSIZE+i]);
//    }
//    DEBUG("\n");



      }

    }
  }


  __syncthreads();

  
}


__global__ void
labelHeavyKernel (int i, int* S,int* SCount,int tw, int* RLoc,int* R, float* dR, float* d,int* vBucketMap){
  //todo: try to increase speed using shared memory for RLoc... think more about it (maybe RLoc is too big for shared memory)
  const unsigned int tid = threadIdx.x;
  const unsigned int num_threads = blockDim.x;

  int mySCount = SCount[0];
  DEBUG("Heavy mySCount %d\n",mySCount);
  for(int k=0; num_threads*k + tid < mySCount;k++){
    int node = S  [ num_threads*k + tid];
    //TODO FIX HEAVY EDGES TEX...
    float downCost   = tex2D(mytex0,node%tw,node/tw);//todo: repeat for left,right and down

    int downRow = node/tw +1;
    int downCol = node%tw;
    int down = downRow*tw+downCol;
    DEBUG("Node %d %f\n", node,down);
    if(down<512*512){
      RLoc[ down ] = num_threads*k + tid;
    
      R[num_threads*k + tid]=-1;
    }
//    dR[num_threads*k + tid]=-1.0;
  }
  DEBUG("done (i=%d) %d\n",i,tid);

  __syncthreads();
  //copy Edges to R
  for(int k=0; num_threads*k + tid < mySCount;k++){
    int node = S  [ num_threads*k + tid];
    //TODO FIX HEAVY EDGES TEX...
    float downCost   = tex2D(mytex0,node%tw,node/tw);//todo: repeat for left,right and down
    //float edgeCost = up;
    int downRow = node/tw +1;
    int downCol = node%tw;
    int down = downRow*tw+downCol;

    if(downCost>DELTA){
      if(down<512*512){
      R[RLoc[down]]  = down;
      dR[RLoc[down]] = d[node]+downCost;
      DEBUG("Heavy new node in R %d (d=%f) from node %d\n",down,dR[RLoc[down]],node);
      vBucketMap[node]=-1;
      }
    }
  }
  
  __syncthreads();
  //todo: empty S here in order to be faster
  
}



__global__ void
emptyKernel(){
}
// B is the bucket i vector
// RLoc[n] stores the position of node n in R (so that if more than one attempt to update
// the distance to node n is made at the same time, it can be shifted to 0,1,2 or 3 in the position of R)


__global__ void
labelBisKernel( int i, int* B,int * Bi,int* R, int * RLoc,float* dR, int* S, int BSize,int* BCount, int* BiCount, int tw , float* d,int* vBucketLoc, int* vBucketMap) {  

 int k;
int v,bn, bn_old;
float x;
int pos;
//  BCount[i]--;
//  *BiCount = 1000;
  //This is the labelling part
  //Firstly, we will mark the position for node n
  //If n appears more than once while expanding Bi, and two threads try to 
  //set different positions for the same node, 
  //it is granted that only one of them will succeed,
  //hence, after the labelling part, each node of Bi will
  //be set to only one place in R
  
  const unsigned int tid = threadIdx.x;
  const unsigned int num_threads = blockDim.x;
//  extern __shared__ 
  __shared__ int Btemp[512];

//  for(int k=0;num_threads * k < BiCount; k++){
//  if(tid<BiCount)//tex2D(tex,2,2))





  //  int Btemptid;

  //Btemp[0]=tex2D(tex,0,0);
  

//  Btemp[tid]=tex2D(tex,node%tw,node/tw);//node%tw,node/tw);//B[tid];

//  __syncthreads();
//  }
//  RLoc[tid]=-1;


  //labelling part finishes here 
  //now we are set to copy edges to R

//for(k=0;k<10;k++){ 
int controlVar = 1;

//while(BCount[0]!=0){

//for(int g=0;g<B2;g++){
//printf("Tid %d Bcount[%d] %d\n",tid,i,BCount[i]);
// controlVar=0;
 if( tid < BCount[i]){
//  printf("Tid %d Bcount[%d] %d\n",tid,i,BCount[i]);

while(BCount[i]!=0){  
//  printf("Tid %d Inside\n",tid,i,BCount[i]);
  int node = B[i*BUCKETSIZE+tid];

//  Btemptid=tex2D(tex,node%tw,node/tw);
//  Btemp[tid]=tex2D(tex,0,0);
//  __syncthreads();
//  if(((node%tw)<1) && ((node%tw)>=0) && ((node/tw)<1) && ((node/tw)>=0))
  int b = node/tw;
  int a = node%tw;
  //WARNING TEX IS WRONG HERE
  Btemp[tid]=tex2D(mytex0,a,b);
  //printf("(tid %d)Tex %d\n",tid,Btemp[tid]);
//__syncthreads();

//printf("Tid %d Still alive\n",tid);

  RLoc[Btemp[tid]]=tid;
//  __syncthreads();

//printf("Tid %d Still alive\n",tid);

  R[RLoc[Btemp[tid]]] = Btemp[tid];
  dR[RLoc[Btemp[tid]]] = d[node]+1.1;

//  __syncthreads();

  //copy B[i] to S
  //duplicates are allowed... they will cause no problem when creating heavy R
  //two threads might be doing the same thing at the same time, which might cause 
  //an access conflit, but that's all

  S[tid] = B[i*BUCKETSIZE+tid];
  

  //TODO: check if clear Bi is correct
  atomicSub(&BCount[i],1);
//  BCount[i]=0;
  controlVar=0;

  
  //Parallel relax light edges
  v = R[tid];
  bn = (int) (dR[tid]/DELTA);
  bn_old = vBucketMap[v];
//  __syncthreads();
//  d[tid]=0.0+bn;
//  d[tid] = __int2float_rn(bn_old)+20;
//  printf("Hello world (%d) bn=%d bn_old=%d\n",tid,bn,bn_old);



  //remove v from old bucket
  if (bn_old != -1) {
    B[bn_old*BSize+vBucketLoc[v]] = GN;
    BCount[bn_old]--;
  }

  //insert v in new bucket
  //TODO: fix BCount
  //  int BCount = 100;
  x = dR[tid];//+BCount[i]+i;

  pos  = BCount[i] + RLoc[v];
  //printf("(tid %d) bn %d pos %d totpos %d v%d\n",tid,bn,pos,bn*BUCKETSIZE+pos,v);
  B[bn*BUCKETSIZE+pos] = v;
  d[v] = x;

  vBucketLoc[v] = pos;  
  vBucketMap[v] = bn;
  RLoc[v]=-1;
  atomicAdd(&BCount[bn],1);
//  BCount[bn]++;


//  __syncthreads();
  }
 }
// }
}

#endif // #ifndef _MEMORY_KERNEL_H_
